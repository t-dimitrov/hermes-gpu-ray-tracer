//#include "Camera.hpp"

#include "hip/hip_runtime.h"


#include <cstdio>

__global__ void CudaPrint()
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    printf("%d\n", i);
}
